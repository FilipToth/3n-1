
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void kernel(double *n, bool *result, double *count) {
    *result = true;

    while (*n != 1)
    {
        // perform modulo
        // a - (a / b) * b => a % b
        if ((*n - (*n / 2) * 2) == 0)
        {
            *n = *n / 2;
        }
        else
        {
            *n = *n * 3;
            *n = *n + 1;
        }

        *count = *count + 1;
    }
}

int main() {
    // Host copies
    double n;
    bool result;
    double count;

    // CUDA Device copies
    double *n_ptr;
    bool *result_ptr;
    double *count_ptr;
    
    hipMalloc(&n_ptr, sizeof(double));
    hipMalloc(&result_ptr, sizeof(bool));
    hipMalloc(&count_ptr, sizeof(double));

    // nums to add
    n = 4;

    // copy to device
    hipMemcpy(n_ptr, &n, sizeof(double1), hipMemcpyHostToDevice);
    
    // start
    kernel<<<1, 1>>>(n_ptr, result_ptr, count_ptr);

    // copy result from device
    hipMemcpy(&result, result_ptr, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(&count, count_ptr, sizeof(double), hipMemcpyDeviceToHost);

    printf("%lf: %d | with %lf operations\n", n, result, count);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }

    return 0;
}