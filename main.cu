
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(int *num1, int *num2, int *result) {
    *result = *num1 + *num2;
}

// 3n+1 problem:
// 
//
//

int main() {
    // Host copies
    int result, num1, num2;

    // CUDA Device copies
    int *result_ptr, *num1_ptr, *num2_ptr;
    
    hipMalloc(&num1_ptr, sizeof(int));
    hipMalloc(&num2_ptr, sizeof(int));
    hipMalloc(&result_ptr, sizeof(int));

    // nums to add
    num1 = 2;
    num2 = 6;

    // copy to device
    hipMemcpy(num1_ptr, &num1, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(num2_ptr, &num2, sizeof(int), hipMemcpyHostToDevice);
    
    // start
    kernel<<<1, 1>>>(num1_ptr, num2_ptr, result_ptr);

    // copy result from device
    hipMemcpy(&result, result_ptr, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", result);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);
    }

    return 0;
}